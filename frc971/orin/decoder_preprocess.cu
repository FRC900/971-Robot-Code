#include "hip/hip_runtime.h"
#include "frc971/orin/decoder_preprocess.h"
#include "frc971/orin/cuda_utils.h"
#include <iostream>

// Kernel to extract stage 2 tag images from full camera image into CH,256,256 tensors

template <typename T, bool isBGR, bool isGray=false, bool threeOutputChannels=true>
__global__ void gpuDecoderPreprocess(const float *H,
									 const T *input, uint32_t iWidth, uint32_t iHeight,
									 float *output, uint32_t oWidth, uint32_t oHeight,
									 float multiplier, float min_value)
{
	const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x >= oWidth) || (y >= oHeight))
		return;

	// Perspective warp
	const float denom = H[6] * x + H[7] * y + H[8];
	const float inputX = (H[0] * x + H[1] * y + H[2]) / denom;
	const float inputY = (H[3] * x + H[4] * y + H[5]) / denom;

	// Bilinear interpolation
	const float cx = max(0.f, min(static_cast<float>(iWidth - 1), inputX));
	const float cy = max(0.f, min(static_cast<float>(iHeight - 1), inputY));

	const int x1 = int(cx);
	const int y1 = int(cy);

	const int x2 = max(0, min(x1 + 1, iWidth - 1));    // bounds check
	const int y2 = max(0, min(y1 + 1, iHeight - 1));

	// compute bilinear weights
	const float x1d = cx - float(x1);
	const float y1d = cy - float(y1);

	const float x1f = 1.0f - x1d;
	const float y1f = 1.0f - y1d;

	const float x2f = 1.0f - x1f;
	const float y2f = 1.0f - y1f;

	const float x1y1f = x1f * y1f;
	const float x1y2f = x1f * y2f;
	const float x2y1f = x2f * y1f;
	const float x2y2f = x2f * y2f;

	const T samples[4] = {
		input[y1 * iWidth + x1],
		input[y1 * iWidth + x2],
		input[y2 * iWidth + x1],
		input[y2 * iWidth + x2]};

	const uint32_t n = oWidth * oHeight; // size of one color channel
	const uint32_t m = y * oWidth + x;	 // pixel offset into channel

	if constexpr (!isGray)
	{
		float3 px{
			samples[0].x * x1y1f + samples[1].x * x2y1f + samples[2].x * x1y2f + samples[3].x * x2y2f,
			samples[0].y * x1y1f + samples[1].y * x2y1f + samples[2].y * x1y2f + samples[3].y * x2y2f,
			samples[0].z * x1y1f + samples[1].z * x2y1f + samples[2].z * x1y2f + samples[3].z * x2y2f
		};

		const float3 rgb = isBGR ? make_float3(px.z, px.y, px.x)
								 : make_float3(px.x, px.y, px.z);

		output[n * 0 + m] = rgb.x * multiplier + min_value;
		output[n * 1 + m] = rgb.y * multiplier + min_value;
		output[n * 2 + m] = rgb.z * multiplier + min_value;
	}
	else
	{
		const float gray = samples[0] * x1y1f + samples[1] * x2y1f + samples[2] * x1y2f + samples[3] * x2y2f;
		output[m] = gray * multiplier + min_value;
		if constexpr (threeOutputChannels)
		{
			output[n + m] = gray * multiplier + min_value;
			output[2 * n + m] = gray * multiplier + min_value;
		}
	}
}
DecoderPreprocess::DecoderPreprocess(void)
{
	cudaSafeCall(hipMalloc(&m_dH, 9 * sizeof(float)));
}
DecoderPreprocess::~DecoderPreprocess(void)
{
	cudaSafeCall(hipFree(m_dH));
}
template <bool isBGR, bool isGray, bool threeOutputChannels>
hipError_t DecoderPreprocess::launchDecoderPreprocess(const float *hH, void *input, imageFormat format, size_t inputWidth, size_t inputHeight,
													   float *output, size_t outputWidth, size_t outputHeight,
													   const float2 &range, hipStream_t stream)
{
	if (!input || !output)
	{
		return hipErrorInvalidDevicePointer;
	}

	if (inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0)
	{
		return hipErrorInvalidValue;
	}

	const float multiplier = (range.y - range.x) / 255.0f;

	// launch kernel
	const dim3 blockDim(16, 16);
	const dim3 gridDim(iDivUp(outputWidth, blockDim.x), iDivUp(outputHeight, blockDim.y));

	cudaSafeCall(hipMemcpyAsync(m_dH, hH, 9 * sizeof(float), hipMemcpyHostToDevice, stream));
	//added BGR options
	if constexpr (!isGray)
	{
		if ((format == imageFormat::IMAGE_RGB8) || (format == imageFormat::IMAGE_BGR8))
			gpuDecoderPreprocess<uchar3, isBGR, isGray><<<gridDim, blockDim, 0, stream>>>(m_dH, (uchar3 *)input, inputWidth, inputHeight, output, outputWidth, outputHeight, multiplier, range.x);
		else if ((format == imageFormat::IMAGE_RGBA8) || (format == imageFormat::IMAGE_BGRA8))
			gpuDecoderPreprocess<uchar4, isBGR, isGray><<<gridDim, blockDim, 0, stream>>>(m_dH, (uchar4 *)input, inputWidth, inputHeight, output, outputWidth, outputHeight, multiplier, range.x);
		else if ((format == imageFormat::IMAGE_RGB32F) || (format == imageFormat::IMAGE_BGR32F))
			gpuDecoderPreprocess<float3, isBGR, isGray><<<gridDim, blockDim, 0, stream>>>(m_dH, (float3 *)input, inputWidth, inputHeight, output, outputWidth, outputHeight, multiplier, range.x);
		else if ((format == imageFormat::IMAGE_RGBA32F) || (format == imageFormat::IMAGE_BGRA32F))
			gpuDecoderPreprocess<float4, isBGR, isGray><<<gridDim, blockDim, 0, stream>>>(m_dH, (float4 *)input, inputWidth, inputHeight, output, outputWidth, outputHeight, multiplier, range.x);
		else
			return hipErrorInvalidValue;
	} 
	else
	{
		// Mono8 option - 1 channel grayscale image, in either uint8_t or float format
		if (format == imageFormat::IMAGE_MONO8)
			gpuDecoderPreprocess<uint8_t, isBGR, isGray, threeOutputChannels><<<gridDim, blockDim, 0, stream>>>(m_dH, (uint8_t *)input, inputWidth, inputHeight, output, outputWidth, outputHeight, multiplier, range.x);
		else if (format == imageFormat::IMAGE_MONO32F)
			gpuDecoderPreprocess<float, isBGR, isGray, threeOutputChannels><<<gridDim, blockDim, 0, stream>>>(m_dH, (float *)input, inputWidth, inputHeight, output, outputWidth, outputHeight, multiplier, range.x);
		else
			return hipErrorInvalidValue;
	}

	return hipGetLastError();
}

// cudaSSDPreprocessRGB
hipError_t DecoderPreprocess::decoderPreprocessRGB(const float *hH, void *input, imageFormat format, size_t inputWidth, size_t inputHeight,
													float *output, size_t outputWidth, size_t outputHeight,
													const float2 &range, hipStream_t stream)
{
	return launchDecoderPreprocess<false, false, false>(hH, input, format, inputWidth, inputHeight, output, outputWidth, outputHeight, range, stream);
}

// cudaSSDPreprocessBGR
hipError_t DecoderPreprocess::decoderPreprocessBGR(const float *hH, void *input, imageFormat format, size_t inputWidth, size_t inputHeight,
													float *output, size_t outputWidth, size_t outputHeight,
													const float2 &range, hipStream_t stream)
{
	return launchDecoderPreprocess<true, false, false>(hH, input, format, inputWidth, inputHeight, output, outputWidth, outputHeight, range, stream);
}

hipError_t DecoderPreprocess::decoderPreprocessGray(const float *hH, void *input, imageFormat format, size_t inputWidth, size_t inputHeight,
													 float *output, size_t outputWidth, size_t outputHeight,
													 const float2 &range, hipStream_t stream)
{
	return launchDecoderPreprocess<false, true, false>(hH, input, format, inputWidth, inputHeight, output, outputWidth, outputHeight, range, stream);
}
hipError_t DecoderPreprocess::decoderPreprocessGrayForThreeChannelOutput(const float *hH, void *input, imageFormat format, size_t inputWidth, size_t inputHeight,
													 float *output, size_t outputWidth, size_t outputHeight,
													 const float2 &range, hipStream_t stream)
{
	return launchDecoderPreprocess<false, true, true>(hH, input, format, inputWidth, inputHeight, output, outputWidth, outputHeight, range, stream);
}