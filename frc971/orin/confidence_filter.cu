#include "hip/hip_runtime.h"
#include <iostream>
#include "frc971/orin/confidence_filter.h"
#include "frc971/orin/cuCompactor.cuh"
#include "frc971/orin/cuda_utils.h"

template <class OUTPUT, class GRID_DIM, class PREDICATE>
ConfidenceFilter<OUTPUT, GRID_DIM, PREDICATE>::ConfidenceFilter(void)
{
    // Output size is calculated as cumulative size up to the last block plus the
    // number of outputs from the final block.
    cudaSafeCall(hipHostMalloc(&m_hOutputCountPtr, sizeof(*m_hOutputCountPtr)));
    cudaSafeCall(hipEventCreate(&m_outputReadyEvent));
}

template <class OUTPUT, class GRID_DIM, class PREDICATE>
ConfidenceFilter<OUTPUT, GRID_DIM, PREDICATE>::~ConfidenceFilter(void)
{
    std::cout << "ConfidenceFilter::~ConfidenceFilter" << std::endl;
    cudaSafeCall(hipFree(m_dOutput));
    cudaSafeCall(hipHostFree(m_hOutputCountPtr));
    cudaSafeCall(hipFree(m_dBlocksCount));
    cudaSafeCall(hipFree(m_dBlocksOffset));

    cudaSafeCall(hipEventDestroy(m_outputReadyEvent));
}

// Set min confidence for a valid element
template <class OUTPUT, class GRID_DIM, class PREDICATE>
ConfidenceFilter<OUTPUT, GRID_DIM, PREDICATE> *ConfidenceFilter<OUTPUT, GRID_DIM, PREDICATE>::withConfidence(const float confidence)
{

    m_predicate.m_confidence = confidence;
    m_needNewCudaGraph = true;

    return this;
}
template <class OUTPUT, class GRID_DIM, class PREDICATE>
bool ConfidenceFilter<OUTPUT, GRID_DIM, PREDICATE>::reallocBuffers(const size_t count, hipStream_t cudaStream)
{
    if (count > m_count)
    {
        cudaSafeCall(hipFreeAsync(m_dOutput, cudaStream));
        cudaSafeCall(hipFreeAsync(m_dBlocksCount, cudaStream));
        cudaSafeCall(hipFreeAsync(m_dBlocksOffset, cudaStream));
        cudaSafeCall(hipMallocAsync(&m_dOutput, sizeof(OUTPUT) * count, cudaStream));

        const uint32_t numBlocks = iDivUp(count, m_blockSize);
        cudaSafeCall(hipMallocAsync(&m_dBlocksCount, sizeof(*m_dBlocksCount) * numBlocks, cudaStream));
        // Add 1 to hold the overall total valid count
        cudaSafeCall(hipMallocAsync(&m_dBlocksOffset, sizeof(*m_dBlocksOffset) * (numBlocks + 1), cudaStream));

        m_count = count;
        return true;
    }
    return false;
}

template <class OUTPUT, class GRID_DIM, class PREDICATE>
void ConfidenceFilter<OUTPUT, GRID_DIM, PREDICATE>::detect(const std::array<const float *, 3> &inputs,
                                                           const GRID_DIM gridDims,
                                                           const float centerVariance,
                                                           const float sizeVariance,
                                                           hipStream_t cudaStream,
                                                           const bool forceCudaGraphRegen)
{
    const bool buffersResized = reallocBuffers(gridDims.size(), cudaStream);
    if (buffersResized || forceCudaGraphRegen || m_needNewCudaGraph)
    {
        std::cout << "Keypoint detector : generating CUDA graph buffersResize = " <<  buffersResized << " forceCudaGraphRegen = " << forceCudaGraphRegen << std::endl;
        cudaSafeCall(hipStreamSynchronize(cudaStream));
        cudaSafeCall(hipStreamBeginCapture(cudaStream, hipStreamCaptureModeGlobal));
        cuCompactor::compact(inputs,
                             m_dOutput,
                             m_hOutputCountPtr,
                             gridDims.data(),
                             centerVariance,
                             sizeVariance,
                             m_count,
                             m_predicate,
                             m_blockSize,
                             m_dBlocksCount,
                             m_dBlocksOffset,
                             cudaStream);
        cudaSafeCall(hipStreamEndCapture(cudaStream, &m_cudaGraph));

        cudaSafeCall(hipGraphInstantiate(&m_cudaGraphInstance, m_cudaGraph, NULL, NULL, 0));
        cudaSafeCall(hipStreamSynchronize(cudaStream));
        m_needNewCudaGraph = false;
    }
    cudaSafeCall(hipGraphLaunch(m_cudaGraphInstance, cudaStream));
    cudaSafeCall(hipEventRecord(m_outputReadyEvent, cudaStream));
}

// Have to wait until a queued async D2H memcpy into m_hOutputCountPtr finishes
// before allowing host to get the number of output which passed filtering
template <class OUTPUT, class GRID_DIM, class PREDICATE>
tcb::span<const OUTPUT> ConfidenceFilter<OUTPUT, GRID_DIM, PREDICATE>::getOutput()
{
    cudaSafeCall(hipEventSynchronize(m_outputReadyEvent));
    return tcb::span<const OUTPUT>(m_dOutput, m_hOutputCountPtr[0]);
}


// Predicate to filter on the max of two confidences
__device__ bool DecoderPredicate::operator()(const float *f, const int index, const int length) const
{
    return max(f[index], f[index + length]) > m_confidence;
}

#include "frc971/orin/grid_prior_value.h"
#include "frc971/orin/stage2_keypoint.h"
template class ConfidenceFilter<Stage2Keypoint, const tcb::span<const GridPriorValue> &, DecoderPredicate>;